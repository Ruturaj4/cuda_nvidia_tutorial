
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 2048*2048 //Total threads
#define M 512 //M is threads/block

__global__ void add(int *a, int *b, int *c, int n)
{

int index = threadIdx.x + blockIdx.x * blockDim.x; //thread_no + block_no * no.ofthreads/block
if (index<n)
c[index] = a[index] + b[index];

}

void random_ints(int* a, int n)
{

int i;
for(i =0; i<n; ++i){
a[i]=rand()/2000;
}

}

int main(void)
{

int *a,*b,*c;
int *d_a, *d_b, *d_c;
int size = N * sizeof(int);

hipMalloc((void**)&d_a, size);
hipMalloc((void**)&d_b, size);
hipMalloc((void**)&d_c, size);

a = (int*)malloc(size);
random_ints(a, N);
b = (int*)malloc(size);
random_ints(b, N);
c = (int*)malloc(size);

hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

add<<<(N+M-1)/M, M>>>(d_a, d_b, d_c, N);

hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

printf("%d + %d is %d\n", *a, *b, *c);

free(a);
free(b);
free(c);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;

}
