
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 512

__global__ void add(int *a, int *b, int *c)
{

c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

}

void random_ints(int* a, int n)
{

int i;
for(i =0; i<n; ++i){
a[i]=rand()/2000;
}

}

int main(void)
{

int *a,*b,*c;
int *d_a, *d_b, *d_c;
int size = N * sizeof(int);

hipMalloc((void**)&d_a, size);
hipMalloc((void**)&d_b, size);
hipMalloc((void**)&d_c, size);

a = (int*)malloc(size);
random_ints(a, N);
b = (int*)malloc(size);
random_ints(b, N);
c = (int*)malloc(size);

hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

add<<<N,1>>>(d_a, d_b, d_c);

hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

printf("%d + %d is %d\n", *a, *b, *c);

free(a);
free(b);
free(c);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;

}
