
#include <hip/hip_runtime.h>
#include "stdio.h"



__global__ void add(int *a, int *b, int *c)
{

*c = *a + *b;

}


int main(void)
{

int a,b,c;
int *d_a, *d_b, *d_c;
int size = sizeof(int);

hipMalloc((void**)&d_a, size);
hipMalloc((void**)&d_b, size);
hipMalloc((void**)&d_c, size);

a=3;
b=4;

hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

add<<<1,1>>>(d_a,d_b,d_c);

hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

printf("%d + %d is %d\n", a, b, c);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;

}
